#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <sys/time.h>

__global__ void
addMatrix( float *C, float *A, float *B, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  int idx = i + j * N;

  if (i < N && j < N)
    C[idx] = A[idx] + B[idx];
}

void call_addMatrix(float *C, float *A, float *B, int N) {
  static int first_time = 1;

  if (first_time) {
    hipSetDevice(cutGetMaxGflopsDeviceId());
    first_time = 0;
  }

  int devID;
  hipDeviceProp_t props;

  cutilSafeCall(hipGetDevice(&devID));
  cutilSafeCall(hipGetDeviceProperties(&props, devID));

  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

  float* d_A;
  cutilSafeCall(hipMalloc((void**) &d_A, sizeof(*A) * N * N));
  float* d_B;
  cutilSafeCall(hipMalloc((void**) &d_B, sizeof(*B) * N * N));

  cutilSafeCall(hipMemcpy(d_A, A, sizeof(*A) * N * N,
                           hipMemcpyHostToDevice) );
  cutilSafeCall(hipMemcpy(d_B, B, sizeof(*B) * N * N,
                           hipMemcpyHostToDevice) );

  float* d_C;
  cutilSafeCall(hipMalloc((void**) &d_C, sizeof(*C) * N * N));

  dim3 dimBlock(16, 16);
  dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);

  addMatrix<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N);

  struct timeval time_start;
  gettimeofday(&time_start, NULL);

  for (int i = 1024; i != 0; --i) {
    addMatrix<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N);
  }

  hipDeviceSynchronize();

  cutilCheckMsg("Kernel execution failed");

  struct timeval time_end;
  gettimeofday(&time_end, NULL);

  double time_float = (time_end.tv_sec - time_start.tv_sec) + (time_end.tv_usec - time_start.tv_usec) * 0.000001;

  printf("CUDA completed 1024 runs in %f\n", time_float);

  cutilSafeCall(hipMemcpy(C, d_C, sizeof(*C) * N * N,
                           hipMemcpyDeviceToHost) );

  cutilSafeCall(hipFree(d_A));
  cutilSafeCall(hipFree(d_B));
  cutilSafeCall(hipFree(d_C));
}

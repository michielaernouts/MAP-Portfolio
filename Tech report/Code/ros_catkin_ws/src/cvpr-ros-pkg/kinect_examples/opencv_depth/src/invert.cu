#include "hip/hip_runtime.h"
#include "invert.cuh"
#define IV_BW 16
#define IV_BH 16

__global__ void invert_kernel
(
	float *in_g,
	float *out_g,
	float  range,
	int    nx,
	int    ny,
	size_t    pitchbytes
)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;


	if(x < nx && y < ny)
	{
		*(float*)((char*)out_g + y*pitchbytes + x*sizeof(float)) = range -
				*(float*)((char*)in_g + y*pitchbytes + x*sizeof(float));
	}
}

__global__ void invert_kernel
(
	float3 *in_g,
	float3 *out_g,
	float  range,
	int    nx,
	int    ny,
	size_t    pitchbytes
)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	float3 result;

	if(x < nx && y < ny)
	{
		result = *(float3*)((char*)in_g + y*pitchbytes + x*sizeof(float3));
		result.x = range - result.x;
		result.y = range - result.y;
		result.z = range - result.z;
		*(float3*)((char*)out_g + y*pitchbytes + x*sizeof(float3)) = result;
	}
}


void invert
(
	float  *in,
	float  *out,
	float  range,
	int    nx,
	int    ny,
	float  **in_out_g,
	size_t *pitchbytes
)
{
	int ngx = (nx%IV_BW) ? ((nx/IV_BW)+1) : (nx/IV_BW);
	int ngy = (ny%IV_BH) ? ((ny/IV_BH)+1) : (ny/IV_BH);
	dim3 dimGrid(ngx,ngy);
	dim3 dimBlock(IV_BW,IV_BH);

	cutilSafeCall( hipMallocPitch((void**)in_out_g,pitchbytes,nx*sizeof(float),ny ));
	cutilSafeCall( hipMemcpy2D(*in_out_g,*pitchbytes,in,nx*sizeof(float),nx*sizeof(float),ny,hipMemcpyHostToDevice));
	invert_kernel<<<dimGrid,dimBlock>>>(*in_out_g,*in_out_g,range,nx,ny,*pitchbytes);
	cutilSafeCall(hipDeviceSynchronize());
	cutilSafeCall( hipMemcpy2D(in,nx*sizeof(float),*in_out_g,*pitchbytes,nx*sizeof(float),ny,hipMemcpyDeviceToHost));
}

void invert
(
	float3 *in,
	float3 *out,
	float  range,
	int    nx,
	int    ny,
	float3 **in_out_g,
	size_t *pitchbytes
)
{
	int ngx = (nx%IV_BW) ? ((nx/IV_BW)+1) : (nx/IV_BW);
	int ngy = (ny%IV_BH) ? ((ny/IV_BH)+1) : (ny/IV_BH);
	dim3 dimGrid(ngx,ngy);
	dim3 dimBlock(IV_BW,IV_BH);

	cutilSafeCall( hipMallocPitch((void**)in_out_g,pitchbytes,nx*sizeof(float3),ny ));
	cutilSafeCall( hipMemcpy2D(*in_out_g,*pitchbytes,in,nx*sizeof(float3),nx*sizeof(float3),ny,hipMemcpyHostToDevice));
	invert_kernel<<<dimGrid,dimBlock>>>(*in_out_g,*in_out_g,range,nx,ny,*pitchbytes);
	cutilSafeCall(hipDeviceSynchronize());
	cutilSafeCall( hipMemcpy2D(in,nx*sizeof(float3),*in_out_g,*pitchbytes,nx*sizeof(float3),ny,hipMemcpyDeviceToHost));
}



void invert
(
	float *in,
	float *out,
	float  range,
	int   nx,
	int   ny,
	float *in_out_g,
	size_t   pitchbytes
)
{
	int ngx = (nx%IV_BW) ? ((nx/IV_BW)+1) : (nx/IV_BW);
	int ngy = (ny%IV_BH) ? ((ny/IV_BH)+1) : (ny/IV_BH);
	dim3 dimGrid(ngx,ngy);
	dim3 dimBlock(IV_BW,IV_BH);

	cutilSafeCall( hipMemcpy2D(in_out_g,pitchbytes,in,nx*sizeof(float),nx*sizeof(float),ny,hipMemcpyHostToDevice));
	invert_kernel<<<dimGrid,dimBlock>>>(in_out_g,in_out_g,range,nx,ny,pitchbytes);
	cutilSafeCall(hipDeviceSynchronize());
	cutilSafeCall( hipMemcpy2D(in,nx*sizeof(float),in_out_g,pitchbytes,nx*sizeof(float),ny,hipMemcpyDeviceToHost));
}

void invert
(
	float3 *in,
	float3 *out,
	float  range,
	int   nx,
	int   ny,
	float3 *in_out_g,
	size_t   pitchbytes
)
{
	int ngx = (nx%IV_BW) ? ((nx/IV_BW)+1) : (nx/IV_BW);
	int ngy = (ny%IV_BH) ? ((ny/IV_BH)+1) : (ny/IV_BH);
	dim3 dimGrid(ngx,ngy);
	dim3 dimBlock(IV_BW,IV_BH);

	cutilSafeCall( hipMemcpy2D(in_out_g,pitchbytes,in,nx*sizeof(float3),nx*sizeof(float3),ny,hipMemcpyHostToDevice));
	invert_kernel<<<dimGrid,dimBlock>>>(in_out_g,in_out_g,range,nx,ny,pitchbytes);
	cutilSafeCall(hipDeviceSynchronize());
	cutilSafeCall( hipMemcpy2D(in,nx*sizeof(float3),in_out_g,pitchbytes,nx*sizeof(float3),ny,hipMemcpyDeviceToHost));
}

void cuda_delete(void *field_g)
{
	cutilSafeCall(hipFree(field_g));
}
